#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <unistd.h>
#include <string.h>
#include "../BGs/isip_ldpc_bg1_i0.h"
#include "../BGs/isip_ldpc_bg1_i1.h"
#include "../BGs/isip_ldpc_bg1_i2.h"
#include "../BGs/isip_ldpc_bg1_i3.h"
#include "../BGs/isip_ldpc_bg1_i4.h"
#include "../BGs/isip_ldpc_bg1_i5.h"
#include "../BGs/isip_ldpc_bg1_i6.h"
#include "../BGs/isip_ldpc_bg1_i7.h"
#include "../BGs/isip_ldpc_bg2_i0.h"
#include "../BGs/isip_ldpc_bg2_i1.h"
#include "../BGs/isip_ldpc_bg2_i2.h"
#include "../BGs/isip_ldpc_bg2_i3.h"
#include "../BGs/isip_ldpc_bg2_i4.h"
#include "../BGs/isip_ldpc_bg2_i5.h"
#include "../BGs/isip_ldpc_bg2_i6.h"
#include "../BGs/isip_ldpc_bg2_i7.h"
#include "util.h"

#define TNPB 35
#define BNPG 1024
#define ITER 5

__constant__ int BG_GPU[46*68];



__global__ 
void BNProcess(int flag, int *BN, int *CN, int *CNbuf, const int *const_llr, int BG_col, int BG_row, int Zc)
{
	int *CNG = (flag)? CN : CNbuf;
	int id = blockIdx.x*blockDim.x + threadIdx.x;

	for(int col = id; col < BG_col*Zc; col += (TNPB*BNPG))
	{
		int tmp = const_llr[col];
		for(int row = 0; row < BG_row; row++)
		{
			int up_shift = (BG_GPU[col/Zc + row*BG_col] - 1)%Zc;
			if(up_shift != -1)
			{
				int row_idx = col%Zc;
				row_idx = row_idx - up_shift;
				if(row_idx < 0)	row_idx = Zc + row_idx;

				row_idx = row*Zc + row_idx;
				tmp = tmp + CNG[row_idx*BG_col*Zc + col];
			}
		}
		BN[col] = tmp;
	}
	__syncthreads();
}



__global__ void CNProcess(int flag, int *BN, int *CN, int *CNbuf, int BG_col, int BG_row, int Zc)
{
	int *CNG	= (flag)? CN : CNbuf;
	int *SCNG 	= (flag)? CNbuf : CN;
	int id = blockIdx.x*blockDim.x + threadIdx.x;
	
	for(int row = id; row < BG_row*Zc; row += (TNPB*BNPG))
	{
		for(int col = 0; col < BG_col; col++)
		{
			int right_shift = BG_GPU[(row/Zc)*BG_col + col] -1;
			if(right_shift != -1)
			{
				int row_idx = row;
//				int col_idx = ((row%384) + right_shift%384) %384 + col*384;
                int col_idx = (row + right_shift) %Zc + col*Zc;
				int sgn_cnt = 0, min = INT32_MAX;
				for(int comp = 0; comp < BG_col; comp++)
				{
					if(comp == col)	continue;
					int comp_right_shift = BG_GPU[(row/Zc)*BG_col + comp] -1;
					if(comp_right_shift != -1)
					{
						int comp_row_idx = row;
//						int comp_col_idx = ((row%384) + (comp_right_shift%384)) %384 + comp*384;
                        int comp_col_idx = (row + comp_right_shift) %Zc + comp*Zc;
						int tmp = BN[comp_col_idx] - CNG[comp_row_idx*BG_col*Zc + comp_col_idx];
						if(tmp < 0)
						{
							tmp = -tmp;
							sgn_cnt++;
						}
						if(tmp < min)	min = tmp;
					}
				}
				SCNG[row_idx*BG_col*Zc + col_idx] = (sgn_cnt%2 == 0)? min: -min;
			}
		}
	}
	__syncthreads();
}

__global__ void BitDetermination(int *BN, unsigned int *decode_d)
{
	__shared__ int tmp[256];
	int tid = blockIdx.x*256 + threadIdx.x;
	int bid = threadIdx.x;
	tmp[bid] = 0;
	
	
	if(BN[tid] < 0)
	{
		tmp[bid] = 1 << (bid&7);
	}

	__syncthreads();
	
	if(threadIdx.x < 32)
	{
		decode_d[blockIdx.x*32 + threadIdx.x] = 0;
		for(int i = 0; i < 8; i++)
		{
			decode_d[blockIdx.x*32 + threadIdx.x] += tmp[threadIdx.x*8+i];
		}
	}
}


// helper function 
void printllr(const char *name, int *src, int *des, int count, int type_size)
{
	cudaCheck( hipMemcpy((void *)des, (const void *)src, count*type_size, hipMemcpyDeviceToHost) );

	FILE *fp = fopen(name, "w");
	if(!fp)	printf("[error]: open file %s failed\n", name);

	for(int i = 0; i < count; i++){
		fprintf(fp, "llr[%d]= %d\n", i, des[i]);
	}
}


int main(int argc, char* argv[])
{
	int opt = 0, block_length = 0, BG = 0, Kb = 0, Zc = 0, BG_row = 0, BG_col = 0, lift_index = 0;
	char file[50] = {};
	short lift_size[51] = {2,3,4,5,6,7,8,9,10,11,12,13,14,15,16,18,20,22,24,26,28,30,32,36,40,44,48,52,56,60,64,72,80,88,96,104,112,120,128,144,160,176,192,208,224,240,256,288,320,352,384};
	short lift_set[][9] = {
		{2,4,8,16,32,64,128,256},
		{3,6,12,24,48,96,192,384},
		{5,10,20,40,80,160,320},
		{7,14,28,56,112,224},
		{9,18,36,72,144,288},
		{11,22,44,88,176,352},
		{13,26,52,104,208},
		{15,30,60,120,240},
		{0}};

	while( (opt = getopt(argc, argv, "l:f:")) != -1){
		switch(opt){
			case 'l':
				block_length = atoi(optarg);
				break;
			case 'f':
				strncpy(file, optarg, strlen(optarg));
				break;
			default:
				fprintf(stderr, "Usage: %s [-l code block length] <-f input file>\n", argv[0]);
				exit(1);
		}
	}
	
	if(block_length == 0 || file[0] == ' '){
		fprintf(stderr, "no input file specified or code block length == 0");
	}

	if(block_length > 3840){
		BG = 1;
		Kb = 22;
		BG_row = 46;
		BG_col = 68;
	}else if(block_length <= 3840){
		BG = 2;
		BG_row = 42;
		BG_col = 52;
		if(block_length > 640)
			Kb = 10;
		else if(block_length > 560)
			Kb = 9;
		else if(block_length > 192)
			Kb = 8;
		else
			Kb = 6;
	}

	for(int i = 0; i < 51; i++){
		if(lift_size[i] >= (double)block_length/Kb){
			Zc = lift_size[i];
			break;
		}
	}
	
	for(int i = 0; lift_set[i][0] != 0; i++){
		for(int j = 0; lift_set[i][j] != 0; j++){
			if(Zc == lift_set[i][j]){
				lift_index = i;
				break;
			}
		}
	}

	int *BG_CPU = NULL;
	switch(lift_index){
		case 0:
			BG_CPU = (BG == 1)? BG1_I0:BG2_I0;
			break;
		case 1:
			BG_CPU = (BG == 1)? BG1_I1:BG2_I1;
			break;
		case 2:
			BG_CPU = (BG == 1)? BG1_I2:BG2_I2;
			break;
		case 3:
			BG_CPU = (BG == 1)? BG1_I3:BG2_I3;
			break;
		case 4:
			BG_CPU = (BG == 1)? BG1_I4:BG2_I4;
			break;
		case 5:
			BG_CPU = (BG == 1)? BG1_I5:BG2_I5;
			break;
		case 6:
			BG_CPU = (BG == 1)? BG1_I6:BG2_I6;
			break;
		case 7:
			BG_CPU = (BG == 1)? BG1_I7:BG2_I7;
			break;
	}
    
//    printf("BG %d lift_index %d Zc %d BG_row %d BG_col %d\n", BG, lift_index, Zc, BG_row, BG_col);
    
	// alloc cpu memory
 	unsigned int *input = (unsigned int*)malloc(sizeof(unsigned int)*8448/8), *decode_output_d, *decode_output_h;
	int *BN, *CN, *CNbuf, *channel_output, *const_llr;
//	debug
//	int *p_BN = (int*)calloc(68*384, sizeof(int));
//	int *p_CN = (int*)calloc(68*384*46*384, sizeof(int));
	int *debug_llr = (int*)calloc(68*384, sizeof(int));
	cudaCheck( hipHostAlloc((void**)&channel_output, 68*384*sizeof(int), hipHostMallocDefault) );
	cudaCheck( hipHostAlloc((void**)&decode_output_h, (8448/8)*sizeof(unsigned int), hipHostMallocMapped) );
	// | hipHostMallocPortable | hipHostMallocMapped | hipHostMallocWriteCombined);
	
	// read data from input file
	ReadDataFromFile(file, input, channel_output, block_length, BG_col, Zc);
	
	// alloc gpu memory 
	// BG
	cudaCheck( hipMemcpyToSymbol(HIP_SYMBOL(BG_GPU), BG_CPU, BG_col*BG_row*sizeof(int)) );
	// LLR CN BN BUF
	size_t p_llr;
	cudaCheck( hipMallocPitch((void**)&const_llr, &p_llr, 68*384*sizeof(int), 1) );
	cudaCheck( hipMallocPitch((void**)&BN, &p_llr, 68*384*sizeof(int), 1) );
	cudaCheck( hipMallocPitch((void**)&CN, &p_llr, 68*384*sizeof(int), 46*384) );
	cudaCheck( hipMallocPitch((void**)&CNbuf, &p_llr, 68*384*sizeof(int), 46*384) );

	cudaCheck( hipMemcpyAsync((void*)const_llr, (const void*)channel_output, 68*384*sizeof(int), hipMemcpyHostToDevice) );
	cudaCheck( hipMemcpyAsync((void*)BN, (const void*)channel_output, 68*384*sizeof(int), hipMemcpyHostToDevice) );
	cudaCheck( hipHostGetDevicePointer((void**)&decode_output_d, (void*)decode_output_h, 0) );
	hipDeviceSynchronize();
	
	hipEvent_t start, end;
	float time;
	
	hipEventCreate(&start);
	hipEventCreate(&end);
	
	hipEventRecord(start,0);

	dim3 grid(BNPG, 1, 1);
	dim3 block(TNPB, 1, 1);
	int flag = 0;
	char str[20] = {};
	for(int it = 0; it < ITER; it++){
		CNProcess<<<grid, block>>>(flag, BN, CN, CNbuf, BG_col, BG_row, Zc);
		flag = (flag+1)&1;
		BNProcess<<<grid, block>>>(flag, BN, CN, CNbuf, const_llr, BG_col, BG_row, Zc);
#ifdef debug 
		snprintf(str, 20,  "%s_%d", "llr", it);
		printllr(str, BN, debug_llr, 68*384, sizeof(int));
#endif
	}
	BitDetermination<<<33, 256>>>(BN, decode_output_d);
	
	hipDeviceSynchronize();
	hipEventRecord(end,0);
	hipEventSynchronize(end);
	hipEventElapsedTime(&time, start, end);

	
	int err_num = 0;
	for(int i = 0; i < block_length/8; i++){
		if(input[i] != decode_output_h[i]){
			printf("input[%d] = %d, decode_output[%d] = %d\n", i, input[i], i, decode_output_h[i]);
			err_num++;
		}
	}
	printf("err_num == %d\n", err_num);
	printf("decode time:%f ms\n",time);
	
	// free resource 
	free(input);
//	free(p_BN);
//	free(p_CN);
	hipHostFree(channel_output);
	hipHostFree(decode_output_h);
	hipFree(const_llr);
	hipFree(BN);
	hipFree(CN);
	hipFree(CNbuf);
	
	return 0;
}
