#include <stdio.h>
#include <hip/hip_runtime.h>
#include "util.h"


void ReadDataFromFile(const char *file, unsigned int *input_data_arr, int *channel_data_arr, int block_length, int BG_col, int Zc)
{
//	static const char testin[] = "../test_input/test_case_1.txt";
	file_t inputfile;
	strcpy(inputfile.filename, file);
	inputfile.fptr = fopen(inputfile.filename, "r");
	if(inputfile.fptr == NULL)
	{	
		puts("cannot open file");
	}
	
	// data processing
	fgets(inputfile.tmp, 100, inputfile.fptr);	// get rid of gen test
	for(int i = 0; i < block_length/8; i++)
	{
		fscanf(inputfile.fptr, "%d", &input_data_arr[i]);
	}
	fgets(inputfile.tmp, 100, inputfile.fptr);	// get rid of '\n'
	fgets(inputfile.tmp, 100, inputfile.fptr);	// get rid of test end
	fgets(inputfile.tmp, 100, inputfile.fptr);	// get rid of channel
	/*
	for(int i = 0; i < 2*384; i++)
	{
		channel_data_arr[i] = 0;
	}
	*/
	for(int i = 0; i < BG_col*Zc; i++)
	{
		fscanf(inputfile.fptr, "%d", &channel_data_arr[i]);
	}
	fclose(inputfile.fptr);
}
