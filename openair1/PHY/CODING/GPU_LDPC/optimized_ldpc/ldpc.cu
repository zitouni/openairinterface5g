#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "bg1_i1_index_array.h"

void print_arr_cpu(const char *file, int *arr, int size)
{
	FILE *fp = fopen(file, "w");
	for(int i = 0; i < size; i++){
		fprintf(fp, "%s[%d]: %d\n", file, i, arr[i]);
	}
	fclose(fp);
}

void print_arr(const char *file, int *arr, int size)
{
	int *tmp = (int*)malloc(sizeof(int)*size);
	FILE *fp = fopen(file, "w");
	hipMemcpy((void*)tmp, (const void*)arr, size*sizeof(int), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	for(int i = 0; i < size; i++){
		fprintf(fp, "%s[%d]: %d\n", file, i, tmp[i]);
	}
	free(tmp);
	fclose(fp);
}

__global__ void llr2CN(float *llr, float *cnbuf, int *l2c_idx)
{
	int tid = blockIdx.x*blockDim.x + threadIdx.x;

	cnbuf[tid] = llr[l2c_idx[tid]];
	__syncthreads();
}

__global__ void llr2BN(float *llr, float *const_llr, int *l2b_idx)
{
	int tid = blockIdx.x*blockDim.x + threadIdx.x;

	const_llr[tid] = llr[l2b_idx[tid]];
	__syncthreads();
}

__global__ void CNProcess(float *cnbuf, float *bnbuf, int *b2c_idx, int *cnproc_start_idx, int *cnproc_end_idx)
{
	int tid = blockIdx.x*blockDim.x + threadIdx.x;

	int start = cnproc_start_idx[tid];
	int end = cnproc_end_idx[tid];
	
	
	int sgn = 1, val = INT32_MAX;
	for(int i = start; i < end; i++){
		if(i == tid)	continue;

		int tmp = cnbuf[i];
		if(tmp < 0){
			tmp = -tmp;
			sgn = -sgn;
		}
		if(val > tmp){
			val = tmp;
		}
	}
	bnbuf[b2c_idx[tid]] = sgn*val;// + const_llr[tid];
	__syncthreads();
}

__global__ void add(int *bnbuf, int start, int pid, int *buf)
{
	__shared__ int sdata[25];
	int tid = threadIdx.x;
	int num = blockDim.x;
	sdata[tid] = bnbuf[start+tid];
	for(int s = num/2; s > 0; s>>=1){
		if(tid < s){
			sdata[tid] += sdata[tid+s];
		}
	}
	if(tid == 0){
		buf[pid] = sdata[tid];
	}
}

__global__ void BNProcess(float *const_llr, float *bnbuf, float *cnbuf, int *c2b_idx, int *bnproc_start_idx, int *bnproc_end_idx, float *resbuf)
{
	int tid = blockIdx.x*blockDim.x + threadIdx.x;
	float val = 0.0;
	
	int start = bnproc_start_idx[tid];
	int end = bnproc_end_idx[tid];
	for(int i = start; i < end; i++){
		if(i == tid)	continue;
		val += bnbuf[i];
	}
	
//	cnbuf[c2b_idx[tid]] = resbuf[tid] + const_llr[tid];
	cnbuf[c2b_idx[tid]] = val + const_llr[tid];
	__syncthreads();
}


__global__ void BN2llr(float *const_llr, float *bnbuf, float *llrbuf, int *llr_idx)
{
	int tid = blockIdx.x*blockDim.x + threadIdx.x;

	int start = llr_idx[tid];
	int end = llr_idx[tid+1];

	int res = 0.0;
	for(int i = start; i < end; i++){
		res += bnbuf[i];
	}
	llrbuf[tid] = res + const_llr[tid];
	__syncthreads();
}

__global__ void BitDetermination(float *BN, unsigned int *decode_d)
{
	__shared__ int tmp[256];
	int tid = blockIdx.x*256 + threadIdx.x;
	int bid = threadIdx.x;
	tmp[bid] = 0;
	
	
	if(BN[tid] < 0)
	{
		tmp[bid] = 1 << (bid&7);
	}

	__syncthreads();
	
	if(threadIdx.x < 32)
	{
		decode_d[blockIdx.x*32 + threadIdx.x] = 0;
		for(int i = 0; i < 8; i++)
		{
			decode_d[blockIdx.x*32 + threadIdx.x] += tmp[threadIdx.x*8+i];
		}
	}
}

void Read_Data(char *filename, int *data_sent, float *data_received)
{
	FILE *fp = fopen(filename, "r");
	fscanf(fp, "%*s");
	for(int i = 0; i < 1056; i++){
		fscanf(fp, "%d", &data_sent[i]);
	}
	fscanf(fp, "%*s");
	fscanf(fp, "%*s");
	fscanf(fp, "%*s");
	for(int i = 0; i < 26112; i++){
		fscanf(fp, "%f", &data_received[i]);
	}
	fclose(fp);
}

int main(int argc, char **argv)
{
	int code_length = 8448, BG = 1;
	int *input = (int*)malloc(1056*sizeof(int));
	float *llr = (float*)malloc(26112*sizeof(float));

	float *llr_d, *llrbuf_d, *const_llr_d, *cnbuf_d, *bnbuf_d, *resbuf_d;
	unsigned int *decode_output_h, *decode_output_d;

	int *l2c_idx_d, *cnproc_start_idx_d, *cnproc_end_idx_d, *c2b_idx_d, *bnproc_start_idx_d, *bnproc_end_idx_d, *b2c_idx_d, *llr_idx_d, *l2b_idx_d;

	char *file = argv[1];
	
	
	int blockNum = 237, threadNum = 512;
	//int blockNum = 33, threadNum = 256;
	//int blockNum = 17, threadNum = 512;

	int rounds = 5, Zc = 384;

	Read_Data(file, input, llr);



	size_t p_llr;
	hipHostAlloc((void**)&decode_output_h, 1056*sizeof(unsigned int), hipHostMallocMapped);

	hipMallocPitch((void**)&llr_d, &p_llr, 26112*sizeof(float), 1);
	hipMallocPitch((void**)&llrbuf_d, &p_llr, 26112*sizeof(float), 1);
	hipMallocPitch((void**)&const_llr_d, &p_llr, 316*384*sizeof(float), 1);
	hipMallocPitch((void**)&cnbuf_d, &p_llr, 316*384*sizeof(float), 1);
	hipMallocPitch((void**)&bnbuf_d, &p_llr, 316*384*sizeof(float), 1);
	hipMallocPitch((void**)&l2c_idx_d, &p_llr, 316*384*sizeof(int), 1);
	hipMallocPitch((void**)&l2b_idx_d, &p_llr, 316*384*sizeof(int), 1);
	hipMallocPitch((void**)&cnproc_start_idx_d, &p_llr, 316*384*sizeof(int), 1);
	hipMallocPitch((void**)&cnproc_end_idx_d, &p_llr, 316*384*sizeof(int), 1);
	hipMallocPitch((void**)&c2b_idx_d, &p_llr, 316*384*sizeof(int), 1);
	hipMallocPitch((void**)&bnproc_start_idx_d, &p_llr, 316*384*sizeof(int), 1);
	hipMallocPitch((void**)&bnproc_end_idx_d, &p_llr, 316*384*sizeof(int), 1);
	hipMallocPitch((void**)&b2c_idx_d, &p_llr, 316*384*sizeof(int), 1);
	hipMallocPitch((void**)&resbuf_d, &p_llr, 316*384*sizeof(int), 1);
	hipMallocPitch((void**)&llr_idx_d, &p_llr, 26113*sizeof(int), 1);

	hipMemcpyAsync((void*)llr_d, (const void*)llr, 68*384*sizeof(float), hipMemcpyHostToDevice);	
	hipMemcpyAsync((void*)l2c_idx_d, (const void*)l2c_idx, 316*384*sizeof(int), hipMemcpyHostToDevice);	
	hipMemcpyAsync((void*)l2b_idx_d, (const void*)l2b_idx, 316*384*sizeof(int), hipMemcpyHostToDevice);	
	hipMemcpyAsync((void*)cnproc_start_idx_d, (const void*)cnproc_start_idx, 316*384*sizeof(int), hipMemcpyHostToDevice);	
	hipMemcpyAsync((void*)cnproc_end_idx_d, (const void*)cnproc_end_idx, 316*384*sizeof(int), hipMemcpyHostToDevice);	
	hipMemcpyAsync((void*)c2b_idx_d, (const void*)c2b_idx, 316*384*sizeof(int), hipMemcpyHostToDevice);	
	hipMemcpyAsync((void*)bnproc_start_idx_d, (const void*)bnproc_start_idx, 316*384*sizeof(int), hipMemcpyHostToDevice);	
	hipMemcpyAsync((void*)bnproc_end_idx_d, (const void*)bnproc_end_idx, 316*384*sizeof(int), hipMemcpyHostToDevice);	
	hipMemcpyAsync((void*)b2c_idx_d, (const void*)b2c_idx, 316*384*sizeof(int), hipMemcpyHostToDevice);	
	hipMemcpyAsync((void*)llr_idx_d, (const void*)llr_idx, 26113*sizeof(int), hipMemcpyHostToDevice);	

	hipHostGetDevicePointer((void**)&decode_output_d, (void*)decode_output_h, 0);
	hipDeviceSynchronize();


	printf("BG %d, Zc %d, code_length %d\n", BG, Zc, code_length);


	hipEvent_t start, end;
	float time;

	hipEventCreate(&start);
	hipEventCreate(&end);
	hipEventRecord(start, 0);


	llr2CN<<<blockNum, threadNum>>>(llr_d, cnbuf_d, l2c_idx_d);
	llr2BN<<<blockNum, threadNum>>>(llr_d, const_llr_d, l2b_idx_d);

/*
	print_arr("debug/const_llr_d", const_llr_d, 26112);
	print_arr("debug/cnbuf_d", cnbuf_d, 316*384);
	print_arr("debug/const_llrbuf_d", const_llrbuf_d, 316*384);
*/

	char dir[] = "debug/", cn[] = "cnbuf", bn[] = "bnbuf", llrstr[] = "llrbuf_d";
	char str[100] = {};
	for(int i = 0; i < rounds; i++){
		CNProcess<<<blockNum, threadNum>>>(cnbuf_d, bnbuf_d, b2c_idx_d, cnproc_start_idx_d, cnproc_end_idx_d);
#ifdef debug
		snprintf(str, 20, "%s%s_%d", dir, bn, i+1);
		print_arr(str, bnbuf_d, 316*384);
#endif
		BNProcess<<<blockNum, threadNum>>>(const_llr_d, bnbuf_d, cnbuf_d, c2b_idx_d, bnproc_start_idx_d, bnproc_end_idx_d, resbuf_d);
#ifdef debug
		snprintf(str, 20, "%s%s_%d", dir, cn, i+1);
		print_arr(str, cnbuf_d, 316*384);
#endif
		BN2llr<<<51, 512>>>(llr_d, bnbuf_d, llrbuf_d, llr_idx_d);
#ifdef debug
		snprintf(str, 20, "%s%s_%d", dir, llrstr, i+1);
		print_arr(str, llrbuf_d, 26112);
#endif
	}

	BitDetermination<<<33, 256>>>(llrbuf_d, decode_output_d);
	hipDeviceSynchronize();


	hipEventRecord(end, 0);
	hipEventSynchronize(end);
	hipEventElapsedTime(&time, start, end);
	printf("time: %.6f ms\n", time);


	int err = 0;
	for(int i = 0; i < 8448/8; i++){
		if(input[i] != decode_output_h[i]){
//			printf("input[%d] :%d, decode_output[%d]: %d\n", i, input[i], i, decode_output_h[i]);
			err++;
		}
	}
	printf("err: %d\n", err);

	free(input);
	free(llr);
	hipFree(llr_d);
	hipFree(llrbuf_d);
	hipFree(bnbuf_d);
	hipFree(cnbuf_d);
	hipFree(l2c_idx_d);
	hipFree(cnproc_start_idx_d);
	hipFree(cnproc_end_idx_d);
	hipFree(c2b_idx_d);
	hipFree(bnproc_start_idx_d);
	hipFree(bnproc_end_idx_d);
	hipFree(b2c_idx_d);
	hipFree(const_llr_d);
	hipFree(llr_idx_d);
	hipFree(resbuf_d);

	hipHostFree(decode_output_h);
	return 0;
}
