#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "test.h"

void print_arr_cpu(const char *file, int *arr, int size)
{
	FILE *fp = fopen(file, "w");
	for(int i = 0; i < size; i++){
		fprintf(fp, "%s[%d]: %d\n", file, i, arr[i]);
	}
	fclose(fp);
}

void print_arr(const char *file, int *arr, int size)
{
	int *tmp = (int*)malloc(sizeof(int)*size);
	FILE *fp = fopen(file, "w");
	hipMemcpy((void*)tmp, (const void*)arr, size*sizeof(int), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	for(int i = 0; i < size; i++){
		fprintf(fp, "%s[%d]: %d\n", file, i, tmp[i]);
	}
	free(tmp);
	fclose(fp);
}

__global__ void llr2CN(int *llr, int *cnbuf, int *l2c_idx)
{
	int tid = blockIdx.x*blockDim.x + threadIdx.x;

	cnbuf[tid] = llr[l2c_idx[tid]];
	__syncthreads();
}

__global__ void llr2BN(int *llr, int *const_llr, int *l2b_idx)
{
	int tid = blockIdx.x*blockDim.x + threadIdx.x;

	const_llr[tid] = llr[l2b_idx[tid]];
	__syncthreads();
}

__global__ void CNProcess(int *cnbuf, int *bnbuf, int *b2c_idx, int *cnproc_idx)
{
	int tid = blockIdx.x*blockDim.x + threadIdx.x;

	int start = cnproc_idx[tid*2];
	int end = cnproc_idx[tid*2+1];
/*	
	if (tid == 6){
		printf("start %d, end %d\n", start, end);
	}
*/
	int sgn = 1, val = INT32_MAX;
	for(int i = start; i < end; i++){
		if(i == tid)	continue;
		
		int tmp = cnbuf[i];
//		if(tid == 6)	printf("tmp %d\n", tmp);
		if(tmp < 0){
			tmp = -tmp;
			sgn = -sgn;
		}
		if(val > tmp){
			val = tmp;
//			if(tid == 6)printf("val = tmp\n");
		}
//		if(tid == 6)	printf("%d\n", val);
	}
/*	
	if(tid == 6){
		printf("b2c_idx[6] %d\n", b2c_idx[tid]);
		printf("sgn*val = %d\n", sgn*val);
		printf("\n\n");
	}
*/
	bnbuf[b2c_idx[tid]] = sgn*val;// + const_llr[tid];
	__syncthreads();
}

__global__ void BNProcess(int *const_llr, int *bnbuf, int *cnbuf, int *c2b_idx, int *bnproc_idx)
{ 
	int tid = blockIdx.x*blockDim.x + threadIdx.x;
	
	int start = bnproc_idx[tid*2];
	int end = bnproc_idx[tid*2+1];
	
	int val = 0;
	for(int i = start; i < end; i++){
		if(i == tid)	continue;
		val += bnbuf[i];
	}
	cnbuf[c2b_idx[tid]] = val + const_llr[tid];
	__syncthreads();
}

__global__ void BN2llr(int *bnbuf, int *llrbuf, int *llr_idx)
{
	int tid = blockIdx.x*blockDim.x + threadIdx.x;

	int start = llr_idx[tid];
	int end = llr_idx[tid+1];

	int res = 0;
	for(int i = start; i < end; i++){
		res += bnbuf[i];
	}
	llrbuf[tid] = res;
	__syncthreads();
}

__global__ void BitDetermination(int *BN, unsigned int *decode_d)
{
	__shared__ int tmp[256];
	int tid = blockIdx.x*256 + threadIdx.x;
	int bid = threadIdx.x;
	tmp[bid] = 0;
	
	
	if(BN[tid] < 0)
	{
		tmp[bid] = 1 << (bid&7);
	}

	__syncthreads();
	
	if(threadIdx.x < 32)
	{
		decode_d[blockIdx.x*32 + threadIdx.x] = 0;
		for(int i = 0; i < 8; i++)
		{
			decode_d[blockIdx.x*32 + threadIdx.x] += tmp[threadIdx.x*8+i];
		}
	}
}


int main(int argc, char **argv)
{
	int llr[15] = {-48, 27, -47, 13, 34, 
				   -41, 51, 29, -6, -19,
				   -45, -42, -40, -6, -33};
	
	int *llr_d, *llrbuf_d, *const_llr_d, *cnbuf_d, *bnbuf_d;
	int *l2c_idx_d, *cnproc_idx_d, *c2b_idx_d, *bnproc_idx_d, *b2c_idx_d, *llr_idx_d, *l2b_idx_d;

	int blockNum = 6, threadNum = 11;
	int rounds = 5, Zc = 3;

	size_t p_llr;

	hipMallocPitch((void**)&llr_d, &p_llr, 15*sizeof(int), 1);
	hipMallocPitch((void**)&llrbuf_d, &p_llr, 15*sizeof(int), 1);
	hipMallocPitch((void**)&const_llr_d, &p_llr, 33*sizeof(int), 1);
	hipMallocPitch((void**)&cnbuf_d, &p_llr, 33*sizeof(int), 1);
	hipMallocPitch((void**)&bnbuf_d, &p_llr, 33*sizeof(int), 1);
	hipMallocPitch((void**)&l2c_idx_d, &p_llr, 33*sizeof(int), 1);
	hipMallocPitch((void**)&l2b_idx_d, &p_llr, 33*sizeof(int), 1);
	hipMallocPitch((void**)&cnproc_idx_d, &p_llr, 33*2*sizeof(int), 1);
	hipMallocPitch((void**)&c2b_idx_d, &p_llr, 33*sizeof(int), 1);
	hipMallocPitch((void**)&bnproc_idx_d, &p_llr, 33*2*sizeof(int), 1);
	hipMallocPitch((void**)&b2c_idx_d, &p_llr, 33*sizeof(int), 1);
	hipMallocPitch((void**)&llr_idx_d, &p_llr, 16*sizeof(int), 1);

	hipMemcpyAsync((void*)llr_d, (const void*)llr, 15*sizeof(int), hipMemcpyHostToDevice);	
	hipMemcpyAsync((void*)l2c_idx_d, (const void*)l2c_idx, 33*sizeof(int), hipMemcpyHostToDevice);	
	hipMemcpyAsync((void*)l2b_idx_d, (const void*)l2b_idx, 33*sizeof(int), hipMemcpyHostToDevice);	
	hipMemcpyAsync((void*)cnproc_idx_d, (const void*)cnproc_idx, 33*2*sizeof(int), hipMemcpyHostToDevice);	
	hipMemcpyAsync((void*)c2b_idx_d, (const void*)c2b_idx, 33*sizeof(int), hipMemcpyHostToDevice);	
	hipMemcpyAsync((void*)bnproc_idx_d, (const void*)bnproc_idx, 33*2*sizeof(int), hipMemcpyHostToDevice);	
	hipMemcpyAsync((void*)b2c_idx_d, (const void*)b2c_idx, 33*sizeof(int), hipMemcpyHostToDevice);	
	hipMemcpyAsync((void*)llr_idx_d, (const void*)llr_idx, 16*sizeof(int), hipMemcpyHostToDevice);	

	hipDeviceSynchronize();



	llr2CN<<<blockNum, threadNum>>>(llr_d, cnbuf_d, l2c_idx_d);
	llr2CN<<<blockNum, threadNum>>>(llr_d, const_llr_d, l2b_idx_d);

/*
	print_arr("debug/const_llr_d", const_llr_d, 26112);
	print_arr("debug/cnbuf_d", cnbuf_d, 316*384);
	print_arr("debug/const_llrbuf_d", const_llrbuf_d, 316*384);
*/

	char debug[] = "debug/";
	char cn[] = "cnbuf";
	char bn[] = "bnbuf";
	char llrstr[] = "llrbuf_d";
	char str[100] = {};
	for(int i = 0; i < rounds; i++){
		CNProcess<<<blockNum, threadNum>>>(cnbuf_d, bnbuf_d, b2c_idx_d, cnproc_idx_d);
		snprintf(str, 20, "%s%s_%d", debug, bn, i+1);
		hipDeviceSynchronize();
		print_arr(str, bnbuf_d, 33);

		BNProcess<<<blockNum, threadNum>>>(const_llr_d, bnbuf_d, cnbuf_d, c2b_idx_d, bnproc_idx_d);
		snprintf(str, 20, "%s%s_%d", debug, cn, i+1);
		print_arr(str, cnbuf_d, 33);

		BN2llr<<<3,5>>>(bnbuf_d, llrbuf_d, llr_idx_d);
		snprintf(str, 20, "%s%s_%d", debug, llrstr, i+1);
		print_arr(str, llrbuf_d, 15);
	}

//	BitDetermination<<<33, 256>>>(llrbuf_d, decode_output_d);
	hipDeviceSynchronize();


	hipFree(llrbuf_d);
	hipFree(bnbuf_d);
	hipFree(cnbuf_d);
	hipFree(l2c_idx_d);
	hipFree(l2b_idx_d);
	hipFree(cnproc_idx_d);
	hipFree(c2b_idx_d);
	hipFree(bnproc_idx_d);
	hipFree(b2c_idx_d);
	hipFree(llr_d);
	hipFree(const_llr_d);
	hipFree(llr_idx_d);

	return 0;
}
